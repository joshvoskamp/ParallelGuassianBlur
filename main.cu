#include "hip/hip_runtime.h"
/*
 * Josh
 * Mario
 *
 * Parallel Gaussian Blur Algorithm
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include "ppmFile.h"

#define max(a, b) ({ \
    __typeof__ (a) _a = (a); __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; \
})
#define min(a, b) ({ \
    __typeof__ (a) _a = (a); __typeof__ (b) _b = (b); \
     _a < _b ? _a : _b; \
})

#define CUDA_CHECK_RETURN(value) {											    \
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);													\
		} }

#define X_Y_TO_INDEX(x, y, width) (((y) * (width) + (x)) * 3)

__global__ void blur(int width, int height, int r, unsigned char* input, unsigned char* output){
    int startRow = blockIdx.x * (height / gridDim.x) + min(blockIdx.x, (height % gridDim.x));

    int endRow = (blockIdx.x + 1) * (height / gridDim.x) + min(blockIdx.x + 1, (height % gridDim.x));

    int pixelsPerSlice = width * (endRow - startRow + 1);

    int blockStartPixel = startRow * width;

    int startPixel = threadIdx.x * (pixelsPerSlice / blockDim.x) + min(threadIdx.x, (pixelsPerSlice % blockDim.x)) + blockStartPixel;

    int endPixel = ((threadIdx.x + 1) * (pixelsPerSlice / blockDim.x) + min(threadIdx.x + 1, (pixelsPerSlice % blockDim.x))) + blockStartPixel;

    for(int k = startPixel; k < endPixel; k++){
            int x = k % width;
            int y = k / width;

            int minX = max((x-r),0);
            int maxX = min((x+r),width-1);

            int minY = max((y-r),0);
            int maxY = min((y+r),height-1);

            int red = 0;
            int green = 0;
            int blue = 0;

            int count = 0;

            for (int i = minX; i<=maxX;i++){
                for(int j = minY; j<=maxY;j++){
                    red = red + input[X_Y_TO_INDEX(i,j,width)+0];
                    green = green + input[X_Y_TO_INDEX(i,j,width)+1];
                    blue = blue + input[X_Y_TO_INDEX(i,j,width)+2];
                    count++;
                }
            }

            red = red/count;
            green = green/count;
            blue = blue/count;

            output[X_Y_TO_INDEX(x,y,width)+0] = red;
            output[X_Y_TO_INDEX(x,y,width)+1] = green;
            output[X_Y_TO_INDEX(x,y,width)+2] = blue;
    }
}

int main(int argc, char* argv[]){
    Image* inputImage;
    int r, width, height;
    char* infile;
    char* outfile;
    unsigned char *d_inputdata;
    unsigned char *d_outputdata;

    int blockSize;
    int minGridSize;

    if (argc < 4){
        printf("Error: Not Enough Arguments\n");
        return 1;
    }

    r = atoi(argv[1]);
    infile = argv[2];
    outfile = argv[3];

    inputImage = ImageRead(infile);

    width = ImageWidth(inputImage);
    height = ImageHeight(inputImage);

    CUDA_CHECK_RETURN(hipMalloc(&d_inputdata, width*height*3*sizeof(unsigned char)));
    CUDA_CHECK_RETURN(hipMalloc(&d_outputdata, width*height*3*sizeof(unsigned char)));
    CUDA_CHECK_RETURN(hipMemcpy(d_inputdata, inputImage->data, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice));

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*) blur, 0, width * height);

    int gridSize = (width * height + blockSize - 1) / blockSize;

    hipEvent_t start, end;
    float elapsedTime;

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&end));

    CUDA_CHECK_RETURN(hipEventRecord(start));
    blur <<< gridSize, blockSize >>> (width, height, r, d_inputdata, d_outputdata);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipEventRecord(end));
    CUDA_CHECK_RETURN(hipEventSynchronize(end));
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, end));


    CUDA_CHECK_RETURN(hipMemcpy(inputImage->data, d_outputdata,width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(d_outputdata));
    CUDA_CHECK_RETURN(hipFree(d_inputdata));

    ImageWrite(inputImage,outfile);
    printf("%s blurred in %f sec\n",infile,(elapsedTime/1000.0));
    return 0;
}
